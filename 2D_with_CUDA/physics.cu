#include "hip/hip_runtime.h"
#include "physics.h"
#include <vector>
#include <string>
#include <hip/hip_cooperative_groups.h>

//TODO:
//  -Try using register hints, 8.6 compute capability should offer me about 80 32-bit registers
//  -Compare Boundary &line = ... performance to Boundary line = ...
//  -When adding the ghost particles for each neighbour, eliminate the extra scopes by using sensible names instead of first_check etc.
//  -When checking whether particle crosses this boundary: first_check<0 or first_check<=0??
//  -if(dens==0.0) dens=0.01; is still very arbitrary
//  -is a for-loop necessary for checking if boundaries_vel is along the average boundary normal?
//  -include your own thread_id when making particle movement changes

// Private declared functions
__global__ void updateParticles(Boundary* boundaries, int numboundaries, Particle* particles, Particle* old_particles, int numpoints, Pump* pumps, PumpVelocity* pumpvelocities, int numpumps, float* pressure_density_ratios);
hipError_t allocateDeviceMemory(Boundary* &device_boundaries, int numboundaries, Particle* &device_particles, Particle* &old_particles, int numpoints, Pump* &device_pumps, PumpVelocity* &device_pumpvelocities, int numpumps, float* &pressure_density_ratios);
void destroyDeviceMemory(Boundary* device_boundaries, int numboundaries, Particle* device_particles, Particle* old_particles, int numpoints, Pump* device_pumps, PumpVelocity* device_pumpvelocities, int numpumps, float* pressure_density_ratios);
hipError_t transferToDeviceMemory(Boundary* boundaries, Boundary* device_boundaries, int numboundaries, Particle* particles, Particle* device_particles, Particle* old_particles, int numpoints, Pump* pumps, Pump* device_pumps, PumpVelocity* pumpvelocities, PumpVelocity* device_pumpvelocities, int numpumps);
hipError_t setupDeviceMemory(Boundary* boundaries, Boundary* &device_boundaries, int numboundaries, Particle* particles, Particle* &device_particles, Particle* &old_particles, int numpoints, Pump* pumps, Pump* &device_pumps, PumpVelocity* pumpvelocities, PumpVelocity* &device_pumpvelocities, int numpumps, float* &pressure_density_ratios);

void physicsBackgroundThread(std::atomic<bool> &exit, std::atomic<bool> &updateRequired, std::atomic<bool> &doneDrawing, Boundary* boundaries, int numboundaries, Particle* particles, int numpoints, Pump* pumps, PumpVelocity* pumpvelocities, int numpumps, HWND m_hwnd){
    Boundary* device_boundaries = NULL;
    Particle* device_particles = NULL;
    Particle* old_particles = NULL;
    Pump* device_pumps = NULL;
    PumpVelocity* device_pumpvelocities = NULL;
    float* pressure_density_ratios = NULL;

    hipError_t success = setupDeviceMemory(boundaries, device_boundaries, numboundaries, particles, device_particles, old_particles, numpoints, pumps, device_pumps, pumpvelocities, device_pumpvelocities, numpumps, pressure_density_ratios);

    while(!exit.load()){
        bool expected = true;
        if(updateRequired.compare_exchange_weak(expected, false)){
            // If an update is necessary, update the particles UPDATES_PER_RENDER times and then redraw the particles
            if(success==hipSuccess){
                dim3 numBlocks((numpoints + BLOCK_SIZE - 1) / BLOCK_SIZE);
                dim3 blockSize(BLOCK_SIZE);
                int sharedMemorySize = numboundaries*sizeof(Boundary)+numpumps*sizeof(Pump)+numpumps*sizeof(PumpVelocity)+SHARED_MEM_PER_THREAD*BLOCK_SIZE;
                //updateParticles<<<numBlocks, BLOCK_SIZE, sharedMemorySize>>>(device_boundaries, numboundaries, device_particles, old_particles, numpoints, device_pumps, device_pumpvelocities, numpumps, pressure_density_ratios);
                void* kernelArgs[] = {&device_boundaries, &numboundaries, &device_particles, &old_particles, &numpoints, &device_pumps, &device_pumpvelocities, &numpumps, &pressure_density_ratios};
                hipLaunchCooperativeKernel(reinterpret_cast<const void*>(updateParticles), numBlocks, blockSize, kernelArgs, sharedMemorySize, 0);
                hipError_t err = hipGetLastError();
                if(DEBUG && err!=hipSuccess){
                    Debugger::updateTopic(DEBUG_THREAD2_ERROR, hipGetErrorString(err));
                }
                while(!doneDrawing.load()){}
                hipMemcpy(particles, device_particles, sizeof(Particle) * numpoints, hipMemcpyDeviceToHost);
            }
            doneDrawing.store(false);

            // Redraw the particles
            InvalidateRect(m_hwnd, NULL, FALSE);
        }
    }

    destroyDeviceMemory(device_boundaries, numboundaries, device_particles, old_particles, numpoints, device_pumps, device_pumpvelocities, numpumps, pressure_density_ratios);
}

__global__ void updateParticles(Boundary* boundaries, int numboundaries, Particle* particles, Particle* old_particles, int numpoints, Pump* pumps, PumpVelocity* pumpvelocities, int numpumps, float* pressure_density_ratios){
    extern __shared__ Boundary s[];

    // Get the grid_group because later on device wide synchronization will be necessary
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();

    // Put all the boundaries and all the pumps in shared memory
    Boundary* boundaries_local_pointer = s;
    Pump* pumps_local_pointer = (Pump*)(&s[numboundaries]);
    PumpVelocity* pumpvelocities_local_pointer = (PumpVelocity*)(&pumps_local_pointer[numpumps]);

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    for(int i=threadIdx.x; i<numboundaries; i+=blockDim.x){
        boundaries_local_pointer[i] = boundaries[i];
    }

    for(int i=threadIdx.x; i<numpumps; i+=blockDim.x){
        pumps_local_pointer[i] = pumps[i];
        pumpvelocities_local_pointer[i] = pumpvelocities[i];
    }

    // Wait until shared memory has been initialized
    __syncthreads();


    float old_x = 0.0;
    float old_y = 0.0;
    float my_x = 0.0;
    float my_y = 0.0;
    if(thread_id < numpoints){ 
        old_x = old_particles[thread_id].x;
        old_y = old_particles[thread_id].y;
        my_x = particles[thread_id].x;
        my_y = particles[thread_id].y;
    }

    for(int e=0; e<UPDATES_PER_RENDER; e++){
        if(thread_id < numpoints){
            float vel_x = (my_x - old_x) / (INTERVAL_MILI/1000.0);
            float vel_y = (my_y - old_y) / (INTERVAL_MILI/1000.0);

            // Update velocities based on whether the particle is in a pump or not
            for(int i = 0; i < numpumps; i++){
                if(my_x >= pumps_local_pointer[i].x_low && my_x <= pumps_local_pointer[i].x_high && my_y >= pumps_local_pointer[i].y_low && my_y <= pumps_local_pointer[i].y_high){
                    vel_x = pumpvelocities_local_pointer[i].velx;
                    vel_y = pumpvelocities_local_pointer[i].vely;
                    break;
                }
            }

            // Update positional change of particles caused by gravity
            vel_y += GRAVITY*PIXEL_PER_METER*(INTERVAL_MILI/1000.0);

            // Update positional change of particles caused boundaries (make sure particles cannot pass boundaries)
            for(int i=0; i<numboundaries; i++){
                Boundary line = boundaries_local_pointer[i];
                float line_nx = line.y2-line.y1;
                float line_ny = line.x1-line.x2;
                float first_check = ((my_x-line.x1)*line_nx+(my_y-line.y1)*line_ny)*((old_x-line.x1)*line_nx+(old_y-line.y1)*line_ny);
                if(first_check > 0) continue;
                float second_check1 = (line.x1-old_x)*line_nx+(line.y1-old_y)*line_ny;
                if(second_check1 < 0) continue;
                float second_check2 = (my_x-old_x)*line_nx+(my_y-old_y)*line_ny;
                float crossing_x = old_x;
                float crossing_y = old_y;
                if(second_check2 > 0.0){
                    crossing_x += (my_x-old_x)*second_check1/second_check2;
                    crossing_y += (my_y-old_y)*second_check1/second_check2;
                }
                float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                if(second_check3>(line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) || second_check4<0.0) continue;
                my_x = crossing_x - (RADIUS/2)*line_nx/sqrtf(line_nx*line_nx+line_ny*line_ny);
                my_y = crossing_y - (RADIUS/2)*line_ny/sqrtf(line_nx*line_nx+line_ny*line_ny);
                vel_x = 0;
                vel_y = 0;
                break;
            }

            // Update particle positions
            old_x = my_x;
            old_y = my_y;
            my_x += vel_x*(INTERVAL_MILI/1000.0);
            my_y += vel_y*(INTERVAL_MILI/1000.0);

            // Also update the particle positions in global memory
            particles[thread_id] = {my_x, my_y};
        }

        // Synchronize the grid
        grid.sync();

        unsigned char* boundary_neighbour_indexes = (unsigned char*)(&pumpvelocities_local_pointer[numpumps])+threadIdx.x*SHARED_MEM_PER_THREAD;
        int number_of_boundary_neighbours = 0;
        unsigned short* particle_neighbour_indexes = NULL;
        int number_of_particle_neighbours = 0;

        float my_pressure_density_ratio = 0.0;

        float boundary_average_nx = 0.0;
        float boundary_average_ny = 0.0;

        if(thread_id < numpoints){
            // Look for boundaries near the particle
            for(unsigned char i=0; i<numboundaries; i++){
                Boundary line = boundaries_local_pointer[i];
                float line_nx = line.y2-line.y1;
                float line_ny = line.x1-line.x2;
                line_nx *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                line_ny *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                float projection = (line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny;
                if(projection >= 0){
                    float crossing_x = my_x + projection*line_nx;
                    float crossing_y = my_y + projection*line_ny;
                    float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                    float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                    bool particle_is_near_to_line = projection <= SMOOTH && second_check3 <= (line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) && second_check4 >= 0;
                    bool particle_is_near_to_endpoint1 = ((line.x1-my_x)*(line.x1-my_x) + (line.y1-my_y)*(line.y1-my_y)) < SMOOTH*SMOOTH && ((line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny) > 0;
                    bool particle_is_near_to_endpoint2 = ((line.x2-my_x)*(line.x2-my_x) + (line.y2-my_y)*(line.y2-my_y)) < SMOOTH*SMOOTH && ((line.x2-my_x)*line_nx+(line.y2-my_y)*line_ny) > 0;
                    if(particle_is_near_to_line || particle_is_near_to_endpoint1 || particle_is_near_to_endpoint2){
                        // Particle is near enough to the boundary
                        boundary_neighbour_indexes[number_of_boundary_neighbours] = i;
                        number_of_boundary_neighbours++;
                        boundary_average_nx += line_nx;
                        boundary_average_ny += line_ny;
                    }
                }
            }

            // Initialize the particle neighbours pointer
            int aligner = (long long)&boundary_neighbour_indexes[number_of_boundary_neighbours] & 1 ? 1 : 0;
            particle_neighbour_indexes = (unsigned short*)(&boundary_neighbour_indexes[number_of_boundary_neighbours]+aligner);

            float dens = 0.0;

            // Find particle neighbours
            for (unsigned short i=0; i < numpoints; i++) {
                Particle p2 = particles[i];
                float dist_squared = (my_x-p2.x)*(my_x-p2.x)+(my_y-p2.y)*(my_y-p2.y);
                if (dist_squared < SMOOTH*SMOOTH) {
                    // If the other particle is close enough, iterate over the closeby boundaries to achieve two things:
                    //  - Convert this particle to a ghost particle over the boundary
                    //  - Check whether the connection between this particle and the particle of the thread crosses a boundary
                    //    in which case the particle is not actually a true neighbour
                    float accumulated_ghost_particle_density = 0.0;
                    int j=0;
                    for(; j<number_of_boundary_neighbours; j++){
                        Boundary line = boundaries_local_pointer[boundary_neighbour_indexes[j]];
                        float line_nx = line.y2-line.y1;
                        float line_ny = line.x1-line.x2;
                        
                        // Check whether particle crosses this boundary
                        {
                            float first_check = ((my_x-line.x1)*line_nx+(my_y-line.y1)*line_ny)*((p2.x-line.x1)*line_nx+(p2.y-line.y1)*line_ny);
                            if(first_check < 0){
                                float second_check1 = (line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny;
                                float second_check2 = (p2.x-my_x)*line_nx+(p2.y-my_y)*line_ny;
                                float crossing_x = my_x;
                                float crossing_y = my_y;
                                if(second_check2 > 0.0){
                                    crossing_x += (p2.x-my_x)*second_check1/second_check2;
                                    crossing_y += (p2.y-my_y)*second_check1/second_check2;
                                }
                                float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                                float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                                if(second_check3<=(line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) && second_check4>=0.0) break;
                            }
                        }

                        // Create a ghost particle over the boundary corresponding to this neighbour
                        {
                            line_nx *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                            line_ny *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                            float projection = (line.x1-p2.x)*line_nx +(line.y1-p2.y)*line_ny;
                            float virtual_x = p2.x + 2*projection*line_nx;
                            float virtual_y = p2.y + 2*projection*line_ny;
                            float first_check = ((my_x-line.x1)*line_nx+(my_y-line.y1)*line_ny)*((virtual_x-line.x1)*line_nx+(virtual_y-line.y1)*line_ny);
                            if(first_check > 0) continue;
                            float second_check1 = (line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny;
                            float second_check2 = (virtual_x-my_x)*line_nx+(virtual_y-my_y)*line_ny;
                            float crossing_x = my_x;
                            float crossing_y = my_y;
                            if(second_check2 > 0.0){
                                crossing_x += (virtual_x-my_x)*second_check1/second_check2;
                                crossing_y += (virtual_y-my_y)*second_check1/second_check2;
                            }
                            float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                            float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                            if(second_check3>(line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) || second_check4<0.0) continue;
                            float dist_squared = (virtual_x-my_x)*(virtual_x-my_x)+(virtual_y-my_y)*(virtual_y-my_y);
                            if(dist_squared > SMOOTH*SMOOTH) continue;
                            float q2 = (float)((1.0 / ((SMOOTH/2)*SQRT_PI))*(1.0 / ((SMOOTH/2)*SQRT_PI))*exp( -dist_squared / (SMOOTH*SMOOTH/4)));
                            accumulated_ghost_particle_density += M_P*q2;
                        }
                    }

                    if(j<number_of_boundary_neighbours) continue;
                    
                    // Change the density caused by ghost particles
                    dens += accumulated_ghost_particle_density;

                    if(i==thread_id) continue;

                    // Change the density because of the neighbour particle and also add the particle to the neighbours list
                    float q2 = (float)((1.0 / ((SMOOTH/2)*SQRT_PI))*(1.0 / ((SMOOTH/2)*SQRT_PI))*exp( -dist_squared / (SMOOTH*SMOOTH/4)));
                    dens += M_P*q2;
                    particle_neighbour_indexes[number_of_particle_neighbours] = i;
                    number_of_particle_neighbours++;
                }
            }
            
            // Make sure no division by zero exceptions occur
            //if(dens<=0.0) dens=0.00001; 

            // Calculate the pressure_density_ratio
            my_pressure_density_ratio = STIFF*(dens-REST)/(dens*dens);
            pressure_density_ratios[thread_id] = my_pressure_density_ratio;
        }

        // Synchronize the grid
        grid.sync();

        if(thread_id < numpoints){
            float vel_x = 0.0;
            float vel_y = 0.0;
            float boundaries_vel_x = 0.0;
            float boundaries_vel_y = 0.0;
            for(int i=0; i<number_of_particle_neighbours; i++){
                unsigned short particle_index = particle_neighbour_indexes[i];
                Particle p2 = particles[particle_index];
                float p2_pressure_density_ratio = pressure_density_ratios[particle_index];
                float press = M_P*(my_pressure_density_ratio + p2_pressure_density_ratio);

                // First calculate displacement of the particle caused by neighbours
                {
                    float dist_squared = (my_x-p2.x)*(my_x-p2.x)+(my_y-p2.y)*(my_y-p2.y);
                    float q = (float)(2*exp( -dist_squared / (SMOOTH*SMOOTH/4)) / (SMOOTH*SMOOTH*SMOOTH*SMOOTH/16) / PI);
                    float displace = (press * q) * (INTERVAL_MILI/1000.0);
                    float abx = (my_x - p2.x);
                    float aby = (my_y - p2.y);
                    vel_x += displace * abx;
                    vel_y += displace * aby;
                }

                // Next calculate displacement of the particle caused by boundaries
                for(int j=0; j<number_of_boundary_neighbours; j++){
                    Boundary line = boundaries_local_pointer[boundary_neighbour_indexes[j]];
                    float line_nx = (line.y2-line.y1);
                    float line_ny = (line.x1-line.x2);
                    line_nx *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                    line_ny *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                    float projection = (line.x1-p2.x)*line_nx +(line.y1-p2.y)*line_ny;
                    float virtual_x = p2.x + 2*projection*line_nx;
                    float virtual_y = p2.y + 2*projection*line_ny;
                    float first_check = ((my_x-line.x1)*line_nx+(my_y-line.y1)*line_ny)*((virtual_x-line.x1)*line_nx+(virtual_y-line.y1)*line_ny);
                    if(first_check > 0) continue;
                    float second_check1 = (line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny;
                    float second_check2 = (virtual_x-my_x)*line_nx+(virtual_y-my_y)*line_ny;
                    float crossing_x = my_x;
                    float crossing_y = my_y;
                    if(second_check2 > 0.0){
                        crossing_x += (virtual_x-my_x)*second_check1/second_check2;
                        crossing_y += (virtual_y-my_y)*second_check1/second_check2;
                    }
                    float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                    float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                    if(second_check3>(line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) || second_check4<0.0) continue;
                    float dist_squared = (virtual_x-my_x)*(virtual_x-my_x)+(virtual_y-my_y)*(virtual_y-my_y);
                    if(dist_squared > SMOOTH*SMOOTH) continue;
                    float q = (float)(2*exp( -dist_squared / (SMOOTH*SMOOTH/4)) / (SMOOTH*SMOOTH*SMOOTH*SMOOTH/16) / PI);
                    float displace = (press * q) * (INTERVAL_MILI/1000.0);
                    float abx = (my_x - virtual_x);
                    float aby = (my_y - virtual_y);
                    boundaries_vel_x += displace * abx;
                    boundaries_vel_y += displace * aby;
                }
            }

            // Also include ghost particles made by the particle itself
            for(int j=0; j<number_of_boundary_neighbours; j++){
                Boundary line = boundaries_local_pointer[boundary_neighbour_indexes[j]];
                float line_nx = (line.y2-line.y1);
                float line_ny = (line.x1-line.x2);
                line_nx *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                line_ny *= 1/sqrtf(line_nx*line_nx+line_ny*line_ny);
                float projection = (line.x1-my_x)*line_nx +(line.y1-my_y)*line_ny;
                float virtual_x = my_x + 2*projection*line_nx;
                float virtual_y = my_y + 2*projection*line_ny;
                float first_check = ((my_x-line.x1)*line_nx+(my_y-line.y1)*line_ny)*((virtual_x-line.x1)*line_nx+(virtual_y-line.y1)*line_ny);
                if(first_check > 0) continue;
                float second_check1 = (line.x1-my_x)*line_nx+(line.y1-my_y)*line_ny;
                float second_check2 = (virtual_x-my_x)*line_nx+(virtual_y-my_y)*line_ny;
                float crossing_x = my_x;
                float crossing_y = my_y;
                if(second_check2 > 0.0){
                    crossing_x += (virtual_x-my_x)*second_check1/second_check2;
                    crossing_y += (virtual_y-my_y)*second_check1/second_check2;
                }
                float second_check3 = (crossing_x-line.x1)*(crossing_x-line.x1)+(crossing_y-line.y1)*(crossing_y-line.y1);
                float second_check4 = (crossing_x-line.x1)*(line.x2-line.x1)+(crossing_y-line.y1)*(line.y2-line.y1);
                if(second_check3>(line.x2-line.x1)*(line.x2-line.x1)+(line.y2-line.y1)*(line.y2-line.y1) || second_check4<0.0) continue;
                float dist_squared = (virtual_x-my_x)*(virtual_x-my_x)+(virtual_y-my_y)*(virtual_y-my_y);
                if(dist_squared > SMOOTH*SMOOTH) continue;
                float q = (float)(2*exp( -dist_squared / (SMOOTH*SMOOTH/4)) / (SMOOTH*SMOOTH*SMOOTH*SMOOTH/16) / PI);
                float displace = (M_P * 2 * my_pressure_density_ratio * q) * (INTERVAL_MILI/1000.0);
                float abx = (my_x - virtual_x);
                float aby = (my_y - virtual_y);
                boundaries_vel_x += displace * abx;
                boundaries_vel_y += displace * aby;
            }

            // Only allow a boundary to cause a velocity change if the particle is repulsed by it 
            // since boundaries should not be able to attract particles
            if(boundary_average_nx*boundaries_vel_x+boundary_average_ny*boundaries_vel_y <=0.0){
                vel_x += boundaries_vel_x;
                vel_y += boundaries_vel_y;
            }

            // Put a velocity limit on the particles too allow the system to work still somewhat normally 
            // if some unforeseen behaviour would occur
            if(vel_x*vel_x+vel_y*vel_y > VEL_LIMIT*VEL_LIMIT){
                vel_x *= VEL_LIMIT/sqrtf(vel_x*vel_x+vel_y*vel_y);
                vel_y *= VEL_LIMIT/sqrtf(vel_x*vel_x+vel_y*vel_y);
            }

            my_x += vel_x*(INTERVAL_MILI/1000.0);
            my_y += vel_y*(INTERVAL_MILI/1000.0);
        }
    }

    grid.sync();

    // Store the both position and old positions in global memeory
    if(thread_id < numpoints){
        old_particles[thread_id] = {old_x, old_y};
        particles[thread_id] = {my_x, my_y};
    }
}

void destroyDeviceMemory(Boundary* device_boundaries, int numboundaries, Particle* device_particles, Particle* old_particles, int numpoints, Pump* device_pumps, PumpVelocity* device_pumpvelocities, int numpumps, float* pressure_density_ratios){
    if(device_boundaries){
        hipFree(device_boundaries);
    }

    if(device_particles){
        hipFree(device_particles);
    }

    if(old_particles){
        hipFree(old_particles);
    }

    if(device_pumps){
        hipFree(device_pumps);
    }

    if(device_pumpvelocities){
        hipFree(device_pumpvelocities);
    }

    if(pressure_density_ratios){
        hipFree(pressure_density_ratios);
    }
}

hipError_t allocateDeviceMemory(Boundary* &device_boundaries, int numboundaries, Particle* &device_particles, Particle* &old_particles, int numpoints, Pump* &device_pumps, PumpVelocity* &device_pumpvelocities, int numpumps, float* &pressure_density_ratios){
    hipError_t status;

    if(numboundaries > 0){
        status = hipMalloc((void**)&device_boundaries, sizeof(Boundary) * numboundaries);
        if(status != hipSuccess){
            return status;
        }
    }

    if(numpoints > 0){
        status = hipMalloc((void**)&device_particles, sizeof(Particle) * numpoints);
        if(status != hipSuccess){
            return status;
        }
        status = hipMalloc((void**)&old_particles, sizeof(Particle) * numpoints);
        if(status != hipSuccess){
            return status;
        }
        status = hipMalloc((void**)&pressure_density_ratios, sizeof(float) * numpoints);
        if(status != hipSuccess){
            return status;
        }
    }

    if(numpumps > 0){
        status = hipMalloc((void**)&device_pumps, sizeof(Pump) * numpumps);
        if(status != hipSuccess){
            return status;
        }
        status = hipMalloc((void**)&device_pumpvelocities, sizeof(PumpVelocity) * numpumps);
        if(status != hipSuccess){
            return status;
        }
    }

    return hipSuccess;
}

hipError_t transferToDeviceMemory(Boundary* boundaries, Boundary* device_boundaries, int numboundaries, Particle* particles, Particle* device_particles, Particle* old_particles, int numpoints, Pump* pumps, Pump* device_pumps, PumpVelocity* pumpvelocities, PumpVelocity* device_pumpvelocities, int numpumps){
    hipError_t status;

    if(numboundaries > 0){
        status = hipMemcpy(device_boundaries, boundaries, sizeof(Boundary) * numboundaries, hipMemcpyHostToDevice);
        if(status != hipSuccess){
            return status;
        }
    }

    if(numpoints > 0){
        status = hipMemcpy(device_particles, particles, sizeof(Particle) * numpoints, hipMemcpyHostToDevice);
        if(status != hipSuccess){
            return status;
        }

        status = hipMemcpy(old_particles, particles, sizeof(Particle) * numpoints, hipMemcpyHostToDevice);
        if(status != hipSuccess){
            return status;
        }
    }

    if(numpumps > 0){
        status = hipMemcpy(device_pumps, pumps, sizeof(Pump) * numpumps, hipMemcpyHostToDevice);
        if(status != hipSuccess){
            return status;
        }

        status = hipMemcpy(device_pumpvelocities, pumpvelocities, sizeof(PumpVelocity) * numpumps, hipMemcpyHostToDevice);
        if(status != hipSuccess){
            return status;
        }
    }

    return hipSuccess;
}

hipError_t setupDeviceMemory(Boundary* boundaries, Boundary* &device_boundaries, int numboundaries, Particle* particles, Particle* &device_particles, Particle* &old_particles, int numpoints, Pump* pumps, Pump* &device_pumps, PumpVelocity* pumpvelocities, PumpVelocity* &device_pumpvelocities, int numpumps, float* &pressure_density_ratios){
    hipError_t success = allocateDeviceMemory(device_boundaries, numboundaries, device_particles, old_particles, numpoints, device_pumps, device_pumpvelocities, numpumps, pressure_density_ratios);
    if(success!=hipSuccess){
        if(DEBUG){
            Debugger::updateTopic(DEBUG_THREAD2_ERROR, ("Physics thread CUDA memory allocations failed, status: "+std::to_string(success)).c_str());
        }
        return success;
    }

    success = transferToDeviceMemory(boundaries, device_boundaries, numboundaries, particles, device_particles, old_particles, numpoints, pumps, device_pumps, pumpvelocities, device_pumpvelocities, numpumps);
    if(success!=hipSuccess){
        if(DEBUG){
            Debugger::updateTopic(DEBUG_THREAD2_ERROR, ("Physics thread CUDA memory transfer failed, status: "+std::to_string(success)).c_str());
        }
        return success;
    }

    // Next, first check whether grid sync is possible by querying the device attribute
    int dev = 0;
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
    if(supportsCoopLaunch!=1){
        if(DEBUG){
            Debugger::updateTopic(DEBUG_THREAD2_ERROR, "CUDA grid synchronization is not supported by the device");
        }
        return hipErrorNotSupported;
    }

    return hipSuccess;
}